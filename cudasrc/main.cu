#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <iostream>
#include <ostream>
#include <sstream>
#include <time.h>

#include <hiprand/hiprand_kernel.h>

#include "raytrace.h"
#include "vec3.h"

void generate_rays(Scene scene, Options option, char *output);


__global__ void ray_generation(vecmath::vec3 *image, CudaScene scene, Options option, hiprandState *random_state)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	float inv_width	 = 1.0f / (float) scene.width;
	float inv_height = 1.0f / (float) scene.height;

	float aspect_ratio = (float) scene.width / (float) scene.height;
	float angle		   = tan(M_PI * 0.5f * option.fov / 180.0f);


	if(x >= scene.width || y >= scene.height)
	{
		return;
	}

	// This was the cpu code:
	float u = (2 * ((x + 0.5) * inv_width) - 1) * angle * aspect_ratio;
	float v = (1 - 2 * ((y + 0.5) * inv_height)) * angle;

	vecmath::vec3 ray_dir;
	ray_dir = scene.camera.direction + u * scene.camera.right + v * scene.camera.up;
	//ray_dir.y -= 1.0f; // this makes the ray_direction correct
	ray_dir = vecmath::normalize(ray_dir);

	Ray ray;
	ray.position  = scene.camera.position;
	ray.direction = ray_dir;
	ray.direction = vecmath::normalize(ray.direction);


	// the pixel index is y * width since width is the max x value, and then this gets us to the y coordinate
	// and then shift over by x amount to hit pixel (x, y)
	int pixel = y * scene.width + x;

	// initialize the random state for this pixel
	hiprand_init(5351 * pixel, 0, 0, &random_state[pixel]);

	image[pixel] = shade(ray, scene, option.max_depth, option.monte_carlo, option.num_path_traces, random_state);
	__syncthreads(); // can't tell if this is necessary; it might be with shared mem access
}


void generate_rays(Scene scene, Options option, char *output)
{
	scene.width	 = 1920;
	scene.height = 1080;

	// The output image that will be written to
	size_t image_size = scene.width * scene.height * sizeof(vecmath::vec3);

	// The image host; will be copied back to CPU to be displayed
	vecmath::vec3 *image_host = (vecmath::vec3 *) malloc(image_size * sizeof(vecmath::vec3));

	// The image device, rendered pixels on gpu
	vecmath::vec3 *image;
	hipMalloc((void **) &image, image_size);

	// Random state to get CUDA RNG
	hiprandState *random_state;
	hipMalloc((void **) &random_state, image_size * sizeof(hiprandState));

	// Copy the Scene over to a CudaScene that lives on the host
	CudaScene host_cuda_scene = CudaScene(scene);

	// The CudaScene that will be passed to the device.
	CudaScene cuda_scene_data = allocate_device_cudascene_struct(host_cuda_scene);



	// Input Binning Starts

	// With camera.position and camera.up & right, we have two planes Ax+By+Cz+D=0, calculate D first
	// A, B, C are just from up and right vectors (they are normal vectors to the above planes)
	float Dup = -1.0 * vecmath::dot(cuda_scene_data.camera.up, cuda_scene_data.camera.position);
	float Dright = -1.0 * vecmath::dot(cuda_scene_data.camera.right, cuda_scene_data.camera.position);
	float lengthup = vecmath::length(cuda_scene_data.camera.up);
	float lengthright = vecmath::length(cuda_scene_data.camera.right);

	// Then, for each sphere, check two things:
	// 1) its relationship with up and right planes;
	// 2) the distance of its center to these two planes vs. its radius;
	// Quadrant 1=top left; 2=top right; 3=bottom left; 4=bottom right;
	// Use prime numbers 2,3,5,7 to index them as a sphere may stay in multiple quadrants;
	for (unsigned int i = 0; i < cuda_scene_data.num_spheres; i++)
	{
		float up = vecmath::dot(cuda_scene_data.camera.up, cuda_scene_data.spheres[i].collider.position) + Dup;
		bool isup = (up >= 0);
		float DistanceToUp = abs(up) / lengthup;
		bool AwayFromUp = (DistanceToUp >= cuda_scene_data.spheres[i].collider.radius);

		float right = vecmath::dot(cuda_scene_data.camera.right, cuda_scene_data.spheres[i].collider.position) + Dright;
		bool isright = (right >= 0);
		float DistanceToRight = abs(right) / lengthright;
		bool AwayFromRight = (DistanceToRight >= cuda_scene_data.spheres[i].collider.radius);

		if(!AwayFromUp)
		{
			if(!AwayFromRight)
			{
				cuda_scene_data.spheres[i].collider.quadrant *= 210; // In quadrant 1,2,3,4;
			}
			else
			{
				if(isright)
				{
					cuda_scene_data.spheres[i].collider.quadrant *= 21; // In quadrant 2,4;
				}
				else
				{
					cuda_scene_data.spheres[i].collider.quadrant *= 10; // In quadrant 1,3;
				}
			}
		}
		else
		{
			if(isup)
			{
				if(!AwayFromRight)
				{
					cuda_scene_data.spheres[i].collider.quadrant *= 6; // In quadrant 1,2;
				}
				else
				{
					if(isright)
					{
						cuda_scene_data.spheres[i].collider.quadrant *= 3; // In quadrant 2;
					}
					else
					{
						cuda_scene_data.spheres[i].collider.quadrant *= 2; // In quadrant 1;
					}
				}
			}
			else
			{
				if(!AwayFromRight)
				{
					cuda_scene_data.spheres[i].collider.quadrant *= 35; // In quadrant 3,4;
				}
				else
				{
					if(isright)
					{
						cuda_scene_data.spheres[i].collider.quadrant *= 7; // In quadrant 4;
					}
					else
					{
						cuda_scene_data.spheres[i].collider.quadrant *= 5; // In quadrant 3;
					}
				}
			}
		}

	}

	// Create arrays for spheres in four quadrants and send them to shared memory later (?);

	// Input Binning Ends

	// Can test different block sizes; this will give us 16 * 16 = 256 = 32 * 8 warps.
	int thread_x = 16;
	int thread_y = 16;

	dim3 blocks;
	blocks.x = scene.width / thread_x + 1;
	blocks.y = scene.height / thread_y + 1;
	blocks.z = 1;

	dim3 grid;
	grid.x = thread_x;
	grid.y = thread_y;
	grid.z = 1;

	// Copy host memory to device memory
	hipMemcpy(image, image_host, image_size, hipMemcpyHostToDevice);

	// Launch kernel
	ray_generation<<<blocks, grid>>>(image, cuda_scene_data, option, random_state);

	// Copy the memory back to the host and then synchronize
	hipMemcpy(image_host, image, image_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	// Read back on the host
	std::ofstream ofs(output, std::ios::out | std::ios::binary);
	ofs << "P6\n"
		<< scene.width << " " << scene.height << "\n255\n";

	for(int i = 0; i < scene.height; i++)
	{
		for(int j = 0; j < scene.width; j++)
		{
			// index from the ray generation kernel
			int index = i * scene.width + j;
			ofs << (unsigned char) (std::min(float(1), image_host[index].x) * 255) << (unsigned char) (std::min(float(1), image_host[index].y) * 255) << (unsigned char) (std::min(float(1), image_host[index].z) * 255);
		}
	}
	ofs.close();
	printf("***\nWROTE TO PPM\n***\n");

	hipFree(image);
}


int main(int argc, char *argv[])
{

	Options option;
	Scene scene;

	int width  = 1024;
	int height = 768;
	char *path;
	char *output;

	bool output_path_passed = false;
	bool path_passed		= false;
	bool use_shadows;

	for(int i = 0; i < argc; i++)
	{
		if(strcmp(argv[i], "--gillum") == 0)
		{
			if(i + 1 < argc)
			{
				option.monte_carlo	   = true;
				option.num_path_traces = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "gillum takes an int after flag for the number of paths traced" << std::endl;
			}
		}

		if(strcmp(argv[i], "--fov") == 0)
		{
			if(i + 1 < argc)
			{
				option.fov = atof(argv[i + 1]);
			}

			else
			{
				std::cerr << "fov takes a float (degrees) after flag for the field of view" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--jsample") == 0)
		{
			if(i + 1 < argc)
			{
				option.grid_size = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "jsample takes an int after flag for the supersampling grid size" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--width") == 0)
		{
			if(i + 1 < argc)
			{
				width = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "width takes an int after flag for the width" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--height") == 0)
		{
			if(i + 1 < argc)
			{
				height = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "height takes an int after flag for the width" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--depth") == 0)
		{
			if(i + 1 < argc && atoi(argv[i + 1]) > 0)
			{
				option.max_depth = atoi(argv[i + 1]);
			}

			else
			{
				std::cerr << "depth takes a positive int after flag for the max depth" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--parallel") == 0)
		{
			if(i + 1 < argc && strcmp(argv[i + 1], "true") == 0)
			{
				option.visual = false;
			}

			if(i + 1 < argc && strcmp(argv[i + 1], "false") == 0)
			{
				option.visual = true;
			}
		}

		if(strcmp(argv[i], "--path") == 0)
		{
			if(i + 1 < argc)
			{
				path		= argv[i + 1];
				path_passed = true;
			}

			else
			{
				std::cerr << "path must be passed after --path" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--output") == 0)
		{
			if(i + 1 < argc)
			{
				output			   = argv[i + 1];
				output_path_passed = true;
			}

			else
			{
				std::cerr << "output path must be passed after --output" << std::endl;
				return 0;
			}
		}

		if(strcmp(argv[i], "--shadow") == 0)
		{
			use_shadows = true;
		}
	}

	if(!path_passed)
	{
		std::cerr << "no scene file was passed. Pass with --path path_to_scn" << std::endl;
		return 0;
	}

	if(!output_path_passed)
	{
		std::cerr << "no output destination was passed. Pass with --output destination_path.ppm" << std::endl;
		return 0;
	}

	scene			  = parseScene(path);
	scene.width		  = width;
	scene.height	  = height;
	scene.use_shadows = use_shadows;

	printf("Above option\n");
	//option.to_string();
	printf("below option\n");
	srand((unsigned) time(0));


	generate_rays(scene, option, output);




	return 0;
}
